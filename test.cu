#include <stdio.h>
#include <chrono>
#include <hip/hip_runtime.h>

#include <iostream>
#include <cudnn.h>
#include <hip/hip_fp16.h>

using namespace std::chrono;

static const char *_cudaGetErrorEnum(hipError_t error) {
  return hipGetErrorName(error);
}

template <typename T>
void check(T result, char const *const func, const char *const file,
           int const line) {
  if (result) {
    fprintf(stderr, "CUDA error at %s:%d code=%d(%s) \"%s\" \n", file, line,
            static_cast<unsigned int>(result), _cudaGetErrorEnum(result), func);
    exit(EXIT_FAILURE);
  }
}

#define checkCudaErrors(val) check((val), #val, __FILE__, __LINE__)


#define checkCUDNN(expression)                             \
{                                                          \
  cudnnStatus_t status = (expression);                     \
  if (status != CUDNN_STATUS_SUCCESS) {                    \
    std::cerr << "Error on line " << __LINE__ << ": "      \
              << cudnnGetErrorString(status) << std::endl; \
    std::exit(EXIT_FAILURE);                               \
  }                                                        \
}

#define checkCUDA(expression)                              \
{                                                          \
  hipError_t status = (expression);                       \
  if (status != hipSuccess) {                             \
    std::cerr << "Error on line " << __LINE__ << ": "      \
              << hipGetErrorString(status) << std::endl;  \
    std::exit(EXIT_FAILURE);                               \
  }                                                        \
}

void print_array(float *array, int size, const char *name) {
  std::cout << name;
  for (int i = 0; i < size; i++) {
    std::cout << array[i] << " ";
  }
  std::cout << std::endl;
}

void init_array(float *array, int size, float val) {
  for (int i = 0; i < size; i++) {
    array[i] = val;
  }
}


void test(){
    hipStream_t stream;
    hipStreamCreate(&stream);

    hipStream_t stream_1;
    hipStreamCreate(&stream_1);

    hipMemPool_t pool_;
    hipDeviceGetDefaultMemPool(&pool_, 0);
    uint64_t threshold = UINT64_MAX;
    hipMemPoolSetAttribute(pool_, hipMemPoolAttrReleaseThreshold, &threshold);

    int _n = 128, _c= 2048, _h = 7, _w = 7;

    int x_size = _n * _c * _h * _w;
    int x_size_bytes = x_size * sizeof(float);
    int iter = 8;
    int mean_size = _c;
    int mean_size_bytes = mean_size * sizeof(float);
    hipGraph_t graph;
    hipGraphExec_t instance;
    bool graphCreated=false;
    float* h_x = (float*)malloc(x_size_bytes);
    float* h_y = (float*)malloc(x_size_bytes);
    init_array(h_x, x_size, 2.5);
    init_array(h_y, x_size, 0.0);
    float *x, *y;
    checkCUDA(hipMalloc(&x, x_size_bytes));
    checkCUDA(hipMalloc(&y, x_size_bytes));
    hipMemcpy(x, reinterpret_cast<const float *>(h_x), x_size_bytes, hipMemcpyHostToDevice);
    float *scale, *offset;
    float *saved_mean, *saved_inv_var;
    float* h_scale = (float*)malloc(mean_size_bytes);
    float* h_offset = (float*)malloc(mean_size_bytes);
    init_array(h_scale, mean_size, 1.5);
    init_array(h_offset, mean_size, 2.0);
    checkCUDA(hipMallocManaged(&scale, mean_size_bytes));
    checkCUDA(hipMallocManaged(&offset, mean_size_bytes));
    hipMemcpy(scale, reinterpret_cast<const float *>(h_scale), mean_size_bytes, hipMemcpyHostToDevice);
    hipMemcpy(offset, reinterpret_cast<const float *>(h_offset), mean_size_bytes, hipMemcpyHostToDevice);
    checkCUDA(hipMallocManaged(&scale, mean_size_bytes));
    checkCUDA(hipMallocManaged(&offset, mean_size_bytes));
    checkCUDA(hipMallocManaged(&saved_mean, mean_size_bytes));
    checkCUDA(hipMallocManaged(&saved_inv_var, mean_size_bytes));
    float *a_x, *a_y, *a_scale, *a_offset, *a_saved_mean, *a_saved_inv_var;
    cudnnHandle_t cudnn;
    checkCUDNN(cudnnCreate(&cudnn));
    for (int i =0; i < iter; i++){
        if (!graphCreated){
          hipStreamBeginCapture(stream, hipStreamCaptureModeGlobal);
          hipMallocFromPoolAsync(reinterpret_cast<hipDeviceptr_t*>(&a_x), x_size_bytes, pool_, stream);
          hipMallocFromPoolAsync(reinterpret_cast<hipDeviceptr_t*>(&a_y), x_size_bytes, pool_, stream);
          hipMallocFromPoolAsync(reinterpret_cast<hipDeviceptr_t*>(&a_scale), mean_size_bytes, pool_, stream);
          hipMallocFromPoolAsync(reinterpret_cast<hipDeviceptr_t*>(&a_offset), mean_size_bytes, pool_, stream);
          hipMallocFromPoolAsync(reinterpret_cast<hipDeviceptr_t*>(&a_saved_mean), mean_size_bytes, pool_, stream);
          hipMallocFromPoolAsync(reinterpret_cast<hipDeviceptr_t*>(&a_saved_inv_var), mean_size_bytes, pool_, stream);
          hipMemcpyAsync(a_x, x, x_size_bytes, hipMemcpyDeviceToDevice, stream);
          hipMemcpyAsync(a_y, y, x_size_bytes, hipMemcpyDeviceToDevice, stream);
          hipMemcpyAsync(a_scale, scale, mean_size_bytes, hipMemcpyDeviceToDevice, stream);
          hipMemcpyAsync(a_offset, offset, mean_size_bytes, hipMemcpyDeviceToDevice, stream);
          hipMemcpyAsync(a_saved_mean, saved_mean, mean_size_bytes, hipMemcpyDeviceToDevice, stream);
          hipMemcpyAsync(a_saved_inv_var, saved_inv_var, mean_size_bytes, hipMemcpyDeviceToDevice, stream);
          auto mode = CUDNN_BATCHNORM_SPATIAL_PERSISTENT;
          float one = 1.0;
          float zero = 0.0;
          //int N = 128, C = 2048, H = 7, W = 7;
          cudnnTensorDescriptor_t x_descriptor;
          checkCUDNN(cudnnCreateTensorDescriptor(&x_descriptor));
          checkCUDNN(cudnnSetTensor4dDescriptor(x_descriptor,
                                                /*format=*/CUDNN_TENSOR_NHWC,
                                                /*dataType=*/CUDNN_DATA_FLOAT,
                                                /*batch_size=*/128,
                                                /*channels=*/2048,
                                                /*image_height=*/7,
                                                /*image_width=*/7));
          cudnnTensorDescriptor_t mean_descriptor;
          checkCUDNN(cudnnCreateTensorDescriptor(&mean_descriptor));
          checkCUDNN(cudnnSetTensor4dDescriptor(mean_descriptor,
                                                /*format=*/CUDNN_TENSOR_NHWC,
                                                /*dataType=*/CUDNN_DATA_FLOAT,
                                                /*batch_size=*/1,
                                                /*channels=*/2048,
                                                /*image_height=*/1,
                                                /*image_width=*/1));
            checkCUDNN(cudnnBatchNormalizationForwardInference(
                  /*handle=*/cudnn,
                  /*mode=*/mode,
                  /*alphaDataDiff=*/&one,
                  /*betaDataDiff=*/&zero,
                  /*xDesc=*/x_descriptor,
                  a_x,
                  /*xDesc=*/x_descriptor,
                  a_y,
                  /*bnScaleBiasMeanVarDesc=*/mean_descriptor,
                  /*bnScale=*/a_scale,
                  /*bnBias=*/a_offset,
                  /*resultSaveMean=*/a_saved_mean,
                  /*resultSaveInvVariance=*/a_saved_inv_var,
                  /*epsilon=*/0.001)
             )

            hipFreeAsync(reinterpret_cast<const hipDeviceptr_t&>(a_x), stream);
            hipFreeAsync(reinterpret_cast<const hipDeviceptr_t&>(a_scale), stream);
            hipFreeAsync(reinterpret_cast<const hipDeviceptr_t&>(a_offset), stream);
            hipFreeAsync(reinterpret_cast<const hipDeviceptr_t&>(a_saved_mean), stream);
            hipFreeAsync(reinterpret_cast<const hipDeviceptr_t&>(a_saved_inv_var), stream);
            checkCudaErrors(hipGraphInstantiate(&instance, graph, NULL, NULL, 0));
            checkCudaErrors(hipGraphUpload(instance, stream));
            graphCreated = true;
        }
        checkCudaErrors(hipGraphLaunch(instance, stream));
        checkCUDA(hipDeviceSynchronize());
        float* out = (float*)malloc(x_size_bytes);
        hipMemcpy(out, reinterpret_cast<const float *>(a_y), x_size_bytes, hipMemcpyDeviceToHost);
        hipFreeAsync(reinterpret_cast<const hipDeviceptr_t&>(a_y), stream);
        print_array(out, x_size, "dx NCHW format: ");
    }

    hipStreamDestroy(stream);
    hipGraphDestroy(graph);
    hipGraphExecDestroy(instance);
    checkCUDA(hipFree(x));
    checkCUDA(hipFree(y));
    checkCUDA(hipFree(scale));
    checkCUDA(hipFree(offset));
    checkCUDA(hipFree(saved_mean));
    checkCUDA(hipFree(saved_inv_var));
    free(h_x);
    free(h_y);
    free(h_scale);
    free(h_offset);
}

int main() {
    test();
    return 0;
}
